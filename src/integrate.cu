#include "hip/hip_runtime.h"
#include "integrate.h"

__global__ 
void integrate_impl(Body* next_state, Body* current_state, int size,
        float time_delta){
    int current_index = blockDim.x*blockIdx.x+ threadIdx.x;
    const Body& this_body = current_state[current_index];

    // the x component and y component of a vector
    Vec2 net_force (0.0, 0.0);
    for(int i = 0; i<size; ++i){
        if(i == current_index)
            continue;

        const Body& other_body = current_state[i];
        float distance = (other_body.pos - this_body.pos).length();
            // G * m1 * m2 / (distance)^3
        float constant_in_front =
            GRAVITATIONAL_CONSTANT *(other_body.mass * this_body.mass) 
                / (distance * distance * distance);
        Vec2 current_force = (other_body.pos - this_body.pos) * constant_in_front;
        net_force = net_force + current_force;
    }

    /// update acceleration 
    next_state[current_index].acc = net_force*(1/this_body.mass);
    next_state[current_index].vel = current_state[current_index].vel + current_state[current_index].acc * time_delta;
    next_state[current_index].pos = current_state[current_index].pos + current_state[current_index].vel * time_delta;
}

// FIXME: next_state, and current_state are from host, 
// we need to perform a memcpy 
void integrate(Body* next_state, Body* current_state, int count, 
        float time_delta){
    Body* next_state_cuda;
    Body* current_state_cuda;
    hipMalloc(&next_state_cuda, count * sizeof(Body));
    hipMemcpy(next_state_cuda, next_state,
            sizeof(Body)*count, hipMemcpyHostToDevice);

    hipMalloc(&current_state_cuda, count * sizeof(Body));
    hipMemcpy(current_state_cuda, current_state,
            sizeof(Body)*count, hipMemcpyHostToDevice);

    // FIXME: we are taking the floor, so there are some objects 
    // that aren't being executed into the buffer
    int block_number =  count / 512;
    // one block and 512 threads, change this later to respect size
    integrate_impl<<<block_number, 512>>>(next_state, current_state, count, time_delta);
    hipDeviceSynchronize();

    hipMemcpy(next_state, next_state_cuda,
            sizeof(Body)*count, hipMemcpyDeviceToHost);
    // FIXME: I don't think we need to copy current state into host device
    hipMemcpy(current_state, current_state_cuda,
            sizeof(Body)*count, hipMemcpyDeviceToHost);
    hipFree(current_state_cuda);
    hipFree(next_state_cuda);
}

